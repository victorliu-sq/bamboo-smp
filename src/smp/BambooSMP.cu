#include "hip/hip_runtime.h"
#include <smp/BambooSMP.cuh>
#include <smp/PRNode.cuh>
#include <utils/utils.hpp>
#include <vector>

namespace bamboosmp {

const int HybridEngine::flag_cpu = 1;
const int HybridEngine::flag_gpu = 2;

HybridEngine::HybridEngine(SMP *smp, const int &thread_limit, const int &size)
    : smp_(smp), n_(size), num_threads_per_block_(thread_limit) {}

// Warm-up kernel function
__global__ void warmUpKernel() {}

void HybridEngine::CheckAndSetupCudaDevices() {
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  if (error_id != hipSuccess) {
    std::cerr << "hipGetDeviceCount returned " << static_cast<int>(error_id)
              << " -> " << hipGetErrorString(error_id) << std::endl;
    std::cerr << "Result = FAIL" << std::endl;
    exit(EXIT_FAILURE);
  }

  if (deviceCount < 2) {
    std::cerr << "At least 2 CUDA devices are required. Found " << deviceCount
              << std::endl;
    exit(EXIT_FAILURE);
  }

  // Set access between device 0 and device 1
  // hipSetDevice(0);
  // hipDeviceEnablePeerAccess(1, 0);
  hipSetDevice(1);
  hipDeviceEnablePeerAccess(0, 0);

  // Run 2 warm-up kernels
  hipSetDevice(1);
  warmUpKernel<<<1, 1>>>();
  hipSetDevice(0);
  warmUpKernel<<<1, 1>>>();

  // Ensure all operations are completed
  hipDeviceSynchronize();
}

void HybridEngine::Solve() {
  Precheck();
  if (!is_perfect_) {
    Init();
    Exec();
    Postproc();
  } else {
    std::cout << "Perfect Case: Skip all subsequent skeps. " << std::endl;
  }
}

auto HybridEngine::GetStableMatching() const -> std::vector<int> {
  std::vector<int> result = std::vector<int>(n_);
  for (int i = 0; i < n_; i++) {
    result[i] = stable_matching_[i];
  }
  return result;
  // return stable_matching_;
}

HybridEngine::~HybridEngine() {
  if (!is_perfect_) {
    CUDA_CHECK(hipFree(device_pref_lists_w_));
    CUDA_CHECK(hipFree(device_pref_lists_m_));
    CUDA_CHECK(hipFree(device_rank_mtx_w_));

    CUDA_CHECK(hipFree(device_next_proposed_w_));
    CUDA_CHECK(hipFree(device_partner_rank_));

    CUDA_CHECK(hipSetDevice(1));
    CUDA_CHECK(hipDeviceDisablePeerAccess(0));
    CUDA_CHECK(hipSetDevice(0));
    CUDA_CHECK(hipDeviceDisablePeerAccess(1));
  }
}

void HybridEngine::PrintMatching() const {
  std::cout << "Stable Matching: " << std::endl;
  for (int i = 0; i < n_; ++i) {
    std::cout << "( Man:" << i
              << " is paired with Woman:" << stable_matching_[i] << ") "
              << std::endl;
  }
  std::cout << std::endl;
}

} // namespace bamboosmp
