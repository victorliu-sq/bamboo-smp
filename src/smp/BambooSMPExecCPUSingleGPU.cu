#include <smp/BambooSMP.cuh>
#include <utils/utils.hpp>

namespace bamboosmp {
    void HybridEngine::doWorkOnCPUSingleGPU() {
        const int threshold = 1;
        const int total = n_ * (n_ - 1) / 2;
        int it = 0;

        int host_free_man_idx;
        int host_num_unproposed;

        // CUDA_CHECK(hipSetDevice(0));

        do {
            host_free_man_idx = total;
            host_num_unproposed = 0;

            CUDA_CHECK(hipMemcpy(temp_host_partner_rank_, device_partner_rank_,
                n_ * sizeof(int), hipMemcpyDeviceToHost));

            for (int w = 0; w < n_; w++) {
                if (temp_host_partner_rank_[w] == n_) {
                    host_num_unproposed++;
                } else {
                    int m_rank = temp_host_partner_rank_[w];
                    host_free_man_idx -= smp_->flatten_pref_lists_w_[w * n_ + m_rank];
                }
            }

            if (host_num_unproposed <= threshold && host_num_unproposed > 0) {
                CUDA_CHECK(hipHostMalloc(&host_prnodes_m_, n_ * n_ * sizeof(PRNode),
                    hipHostMallocDefault));

                CUDA_CHECK(hipMemcpy(host_prnodes_m_, device_prnodes_m_,
                    n_ * n_ * sizeof(PRNode), hipMemcpyDeviceToHost));

                if (atomic_host_terminate_flag_.load() == 0) {
                    LAProcedure(host_free_man_idx);

                    int expected = 0;
                    if (atomic_host_terminate_flag_.compare_exchange_strong(expected,
                                                                            flag_cpu)) {
                        std::cout << "CheckKernel has won the contention" << std::endl;
                        int host_terminate_flag = atomic_host_terminate_flag_.load();
                    }
                }
                host_num_unproposed = 0;
            }
            it++;
        } while (host_num_unproposed != 0);
    }
} // namespace bamboosmp
